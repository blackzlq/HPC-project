#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

#define PI 3.14159265
__host__ void readdata(double **ArrayX,double **ArrayF,double *Arrayq)
{
    
    
}
__device__ double sinc(double x)
{
	if(x==0)
		return 1;
	else return sin(x)/x;
}
//since max number of threads is 1024 per block which is smaller than 214*214
//so separate the calculation,214 x blocks and 214 threads for each of 60 y blocks
//by which I can use shared memory to do reducation
 
__global__ void cal_output_first(double *ArrayF,double *Arrayq,double *dist,double *output_first)
{
    int i,j,k;
    i=threadIdx.x;
    j=blockIdx.x;
    k=blockIdx.y;
    __shared__ double sdata[214];

    int tid=threadIdx.x;
	if(threadIdx.x<214&&blockIdx.y<60&&blockIdx.x<214)
{
	sdata[tid]=ArrayF[i+214*k]*ArrayF[j+214*k]*sinc(Arrayq[k]*dist[i+214*j]);
    __syncthreads();
    double sum=0;

    for(int m=0;m<214;m++)
    {
        sum+=sdata[m];
        __syncthreads();
    }
    if(tid==0)
	output_first[blockIdx.x+blockIdx.y*blockDim.x]=sum;
}
}
//calculate sum of remaining 214 value for outputq[60]
//214 threads and 60 x blocks
__global__ void cal_output_second(double *output_first,double *outputQ)
{
    int i,k;
    i=threadIdx.x;
    k=blockIdx.x;
    __shared__ double sdata[214]; 
if(threadIdx.x<214&&blockIdx.x<60)
{
    int tid=threadIdx.x;
        sdata[tid]=output_first[i+214*k];
    __syncthreads();
    double sum=0;

    for(int m=0;m<214;m++)
    {
        sum+=sdata[m];
        __syncthreads();
    }
    if(tid==0)
        outputQ[blockIdx.x]=sum;
}
}
__global__ void cal_dist(double *ArrayX,double *dist)
{
    int i,j;
    i=threadIdx.x;
    j=blockIdx.x;
if(i<214&&j<214)
{
    double temp1,temp2,temp3;
    temp1=(ArrayX[i]-ArrayX[j])*(ArrayX[i]-ArrayX[j]);
    temp2=(ArrayX[i+214]-ArrayX[j+214])*(ArrayX[i+214]-ArrayX[j+214]);
    temp3=(ArrayX[i+428]-ArrayX[j+428])*(ArrayX[i+428]-ArrayX[j+428]);
	
    dist[i+214*j]=sqrt(temp1+temp2+temp3);

//printf("temp is %d",temp);
}
}
int main()
{
    hipEvent_t start = 0;
    hipEvent_t stop = 0;
    float time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    double ArrayF[214*60],Arrayq[60],ArrayX[214*3],outputQ[60];
    int i,j;
    int sizeF=214*60,sizeq=60,sizeX=214*3;
    int sizeQ=60,sizedist=214*214;
    ifstream F("dataF.txt"),q("dataq.txt"),X("dataX.txt");
   
    for(j=0;j<3;j++)
        for(i=0;i<214;i++)
        {
            X>>ArrayX[i+j*214];
        }
    for(j=0;j<60;j++)
        for(i=0;i<214;i++)
        {
            
            F>>ArrayF[i+214*j];
        }
    
    for(j=0;j<60;j++)
    {
        q>>Arrayq[j];
        //cout<<Arrayq[1][j]<<endl;
    }
    //calculate distance dist[214*214]
    int dsizeX=sizeX*sizeof(double);
    int dsizedist=sizedist*sizeof(double);
    int dsizeF=sizeF*sizeof(double);
    int dsizeq=sizeq*sizeof(double);
    int dsizeQ=sizeQ*sizeof(double);
    int dsize_first=214*60*sizeof(double);
    double *d_dist,*d_ArrayX,*d_ArrayF,*d_Arrayq,*d_outputQ;
    double *d_output_first;
    //hipEventRecord(start,0);
    hipMalloc((void**)&d_dist,dsizedist);
    hipMalloc((void**)&d_ArrayX,dsizeX);
    hipMalloc((void**)&d_ArrayF,dsizeF);
    hipMalloc((void**)&d_Arrayq,dsizeq);
    hipMalloc((void**)&d_outputQ,dsizeQ);
    //allocate memory for output_first
    hipMalloc((void**)&d_output_first,dsize_first);
    hipMemcpy(d_ArrayX,&ArrayX,dsizeX,hipMemcpyHostToDevice);
    hipMemcpy(d_ArrayF,&ArrayF,dsizeF,hipMemcpyHostToDevice);
    hipMemcpy(d_Arrayq,&Arrayq,dsizeq,hipMemcpyHostToDevice);
    hipMemcpy(d_outputQ,&outputQ,dsizeQ,hipMemcpyHostToDevice);
    dim3 DimGrid1(256,1,1);
    dim3 DimBlock1(256, 1,1);
    dim3 DimGrid2(256,64,1);
    dim3 DimBlock2(256, 1,1);
    dim3 DimGrid3(64,1,1);
    dim3 DimBlock3(256, 1,1);
    hipEventRecord(start,0);
    cal_dist<<<DimGrid1,DimBlock1>>>(d_ArrayX,d_dist);
    //hipEventRecord(stop,0);
    //hipEventSynchronize(stop);
    hipDeviceSynchronize();
    cal_output_first<<<DimGrid2,DimBlock2>>>(d_ArrayF,d_Arrayq,d_dist,d_output_first);
    //output_first size 214*60
    //calculate output
    cal_output_second<<<DimGrid3,DimBlock3>>>(d_output_first,d_outputQ);
    //hipMemcpy(&outputQ,d_outputQ,dsizeQ,hipMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipMemcpy(&outputQ,d_outputQ,dsizeQ,hipMemcpyDeviceToHost);
    hipFree(d_dist);
    hipFree(d_ArrayX);
    hipFree(d_ArrayF);
    hipFree(d_Arrayq);
    hipFree(d_outputQ);
    //Free memory for output_first
    hipFree(d_output_first);

    for(i=0;i<60;i++)
    {
	cout<<outputQ[i]<<endl;
    }
    /*
    for(j=0;j<214;j++)
	{
		for(i=0;i<214;i++)
	cout<<i<<","<<j<<"__"<<dist[i+j*214]<<endl;
	cout<<endl;
	}
    */
    	hipEventElapsedTime(&time,start,stop);
        cout<<"Time for the kernel: "<<time<<endl;
    return EXIT_SUCCESS;
     
}

