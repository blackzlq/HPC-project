#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

#define PI 3.14159265
double sinc(double x);
__host__ void inital(float *v1,float *v2,int vsize)
{
	srand(time(NULL));
	int i;
	for(i=0;i<vsize;i++)
	{
		v1[i]=static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/10));	

		v2[i]=static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/10));
		//cout<<v1[i]<<"--"<<v2[i]<<endl;
	}
}
__global__ void VecAdd(float *v1,float *v2,float *v3,int n)
{
	int i = threadIdx.x+blockDim.x*blockIdx.x+blockDim.x*blockDim.x*blockIdx.y;
	//cout<<i<<endl;
	if(i<n)
	{
		v3[i]=v1[i]+v2[i];
	}
}

/*
int VecAdd(float *v1,float *v2,float *v3,int n)
{
	// Run ceil(n/1000) blocks of 1000 threads each
	dim3 DimGrid(ceil(n/1000.0), 1, 1);
	dim3 DimBlock(1000, 1, 1);
	VecAdd<<<DimGrid,DimBlock>>>(v1, v2, v3);	
}


__device__ float compute(float *v1,float *v2)
{

}
*/
int main()
{
	hipEvent_t start = 0;
	hipEvent_t stop = 0;
	float time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	double TestSin[60],ArrayF[214][60],Arrayq[60],ArrayX[214][3],outputQ[60],dist[214][214];

	//initialize vector
	inital(v1,v2,vsize);
	//check data
	/*
	for(i=0;i<10;i++)
	{
		cout<<v1[i]<<"--"<<v2[i]<<endl;
	}
	*/
 	//allocate memory 1000 threads per block
	int fsize = vsize*sizeof(float);
	
	float *v1_d,*v2_d,*v3_d;
	hipMalloc((void**)&v1_d,fsize);
	hipMemcpy(v1_d,v1,fsize,hipMemcpyHostToDevice);
	hipMalloc((void**)&v2_d,fsize);
        hipMemcpy(v2_d,v2,fsize,hipMemcpyHostToDevice);
	hipMalloc((void**)&v3_d,fsize);
        
	//hipMemcpy(v3_d,v3,fsize,hipMemcpyHostToDevice);

	//kernel code
	//int bsize=vsize/1000;
	int gridsizex=ceil(vsize/1024.0);
	if(ceil(vsize/1024.0)>1024)
		gridsizex = 1024;
	
	int gridsizey = ceil(vsize/1024.0/1024.0);
	//cout<<gridsizex<<endl;
	dim3 DimGrid(gridsizex,gridsizey,1);
	dim3 DimBlock(1024, 1,1);
	hipEventRecord(start,0);
	VecAdd<<<DimGrid,DimBlock>>>(v1_d,v2_d,v3_d,vsize);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	//Synchronize
	hipDeviceSynchronize();
	//Transfer v3 back to host
	hipMemcpy(v3,v3_d,fsize,hipMemcpyDeviceToHost);
	
	//output
	//check	
	for(i=0;i<vsize;i++)
	{
		if(v3[i]!=v1[i]+v2[i])
		{
			cout<<i<<"--"<<v1[i]<<"--"<<v2[i]<<"--"<<v3[i]<<endl;
			break;
		}
	}
	
	//Free memory
	hipFree(v1_d);hipFree(v2_d);hipFree(v3_d);

	hipEventElapsedTime(&time,start,stop);
	cout<<"Time for the kernel: "<<time<<endl;
	return EXIT_SUCCESS;
}
